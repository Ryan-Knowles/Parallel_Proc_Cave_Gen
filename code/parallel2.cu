#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <Windows.h>

#define DO_STATS
#define M_SIZE		1024
typedef struct {
	int births;
	int deaths;
	int alive;
	int dead;
} MapStats;

template <size_t xSize, size_t ySize, size_t zSize>
void initMap(unsigned char *oldMap, int mapSeed);
template <size_t xSize, size_t ySize, size_t zSize>
void printJSON(unsigned char *map, int iter);
template <size_t xSize, size_t ySize, size_t zSize>
void countStats(unsigned char *oldMap, unsigned char *newMap, MapStats &stats);

//On Device function
template <size_t xSize, size_t ySize, size_t zSize>
__device__ int countNeighbours(unsigned char *map, int x, int y, int z);
//Kernels
template<size_t xSize, size_t ySize, size_t zSize>
__global__ void unopIterate(unsigned char *d_oldMap, unsigned char *d_newMap, int iters, int bLim, int dLim);


//Globals -- bad code
const int deathLimit = 15;
const int birthLimit = 17;

int main()
{
	const int mapSeed = 45000;
	const int xSize = M_SIZE;
	const int ySize = M_SIZE;
	const int zSize = M_SIZE;
	const int maxIters = 30;
	const bool TIME = true;
	
	const int mapSize = xSize*ySize*zSize;
	
	const int blockSize = 8;
	dim3 blockDim(blockSize, blockSize, blockSize);
	dim3 gridDim(xSize/blockSize, ySize/blockSize, zSize/blockSize);
	
	clock_t start, total = 0;
	unsigned char *oldMap = new unsigned char[xSize*ySize*zSize];
	unsigned char *newMap = new unsigned char[xSize*ySize*zSize];
	unsigned char *temp;
	MapStats stats;
	
	unsigned char *d_oldMap;
	unsigned char *d_newMap;
	
	hipMalloc((void **) &d_oldMap, mapSize);
	hipMalloc((void **) &d_newMap, mapSize);
	
	initMap<xSize,ySize,zSize>(oldMap, mapSeed);
			
	#ifdef DO_STATS
	
	#endif
	
	//Main iteration section
	if(TIME) start = clock();
	hipMemcpy(d_oldMap, oldMap, mapSize, hipMemcpyHostToDevice);
	for(int iter=0; iter<maxIters; ++iter)
	{		
		unopIterate<xSize,ySize,zSize><<<gridDim,blockDim>>>(d_oldMap, d_newMap, 1, birthLimit, deathLimit);
//		if(TIME) total += clock() - start;
//		Sleep(50);
//		printf("[%d] ", iter+1);
//		if(TIME) start = clock();
		if(iter != maxIters-1)
		{
			temp = d_oldMap;
			d_oldMap = d_newMap;
			d_newMap = temp;
		}
	}
	hipMemcpy(newMap, d_newMap, mapSize, hipMemcpyDeviceToHost);
	if(TIME) total += clock() - start;
	//printf("\n");
	hipMemcpy(oldMap, d_oldMap, mapSize, hipMemcpyDeviceToHost);
	#ifdef DO_STATS
	printf("[%d] ",maxIters);
	countStats<xSize,ySize,zSize>(oldMap, newMap, stats);
	printf("births: %d   \tdeaths: %d   \talive: %d   \tdead: %d   \ttotal: %d\n", stats.births, stats.deaths, stats.alive, stats.dead, stats.alive+stats.dead);
	#endif

	if(TIME)
	{
		double diff = (double(total))/CLOCKS_PER_SEC;
		printf("time: took %f seconds for %dx%dx%d matrix\n", diff, xSize, ySize, ySize);
	}
	
	delete[] oldMap;
	delete[] newMap;
	
	return 0;
}

template <size_t xSize, size_t ySize, size_t zSize>
void initMap(unsigned char *oldMap, int mapSeed)
{
	srand(mapSeed);
	for(int k=0; k<zSize; ++k)
	{
		for(int j=0; j<ySize; ++j)
		{
			for(int i=0; i<xSize; ++i)
			{
				oldMap[k*(xSize*ySize)+j*xSize+i] = rand() % 2;
			}
		}
	}
}

//Count neighbours that are alive
template <size_t xSize, size_t ySize, size_t zSize>
__device__ int countNeighbours(unsigned char *map, int x, int y, int z)
{
	const bool countBounds = true;
    int count = 0;
    for(int k=-1; k<2; ++k)
	{
        for(int j=-1; j<2; ++j)
		{
			for(int i=-1; i<2; ++i)
			{
				//Count all except middle point
				if( i != 0 || j != 0 || k != 0)
				{
					int xPos = x + i;
					int yPos = y + j;
					int zPos = z + k;
					
					//Check boundaries
					if(xPos < 0 || yPos < 0 || zPos < 0 || xPos >= xSize || yPos >= ySize || zPos >= zSize)
					{
						//if(x==0 && y==0 && z==0) printf("(%d,%d,%d):bounds\n",xPos,yPos,zPos);
						if(countBounds) count++;
					}
					else 
					{
						//if(x==0 && y==0 && z==0) printf("(%d,%d,%d):not bounds\n",xPos,yPos,zPos);
						count += map[zPos*(xSize*ySize)+yPos*xSize+xPos];
					}
				}
			}
        }
    }
	return count;
}

template<size_t xSize, size_t ySize, size_t zSize>
__global__ void unopIterate(unsigned char *d_oldMap, unsigned char *d_newMap, int iters, int bLim, int dLim)
{
	const int globalx = (blockIdx.x * blockDim.x) + threadIdx.x;
	const int globaly = (blockIdx.y * blockDim.y) + threadIdx.y;
	const int globalz = (blockIdx.z * blockDim.z) + threadIdx.z;
	
	//Only perform action if thread is inside the bounds of the grid
	if( !(globalx >= xSize || globaly >= ySize || globalz >= zSize) )
	{
		int globalIndex = globalz*(xSize * ySize) + globaly*(xSize)+globalx;
		int aliveCnt = countNeighbours<xSize,ySize,zSize>(d_oldMap, globalx, globaly, globalz);
		if(d_oldMap[globalIndex] == 1) 
		{
			d_newMap[globalIndex] = (aliveCnt < dLim) ? 0 : 1;
		}
		else
		{
			d_newMap[globalIndex] = (aliveCnt > bLim) ? 1 : 0;
		}
	}
}


template <size_t xSize, size_t ySize, size_t zSize>
void printJSON(unsigned char *map, int iter)
{
	printf("\t{\n");
	printf("\t\"iteration\" : %d,\n", iter);
	printf("\t\"map\" : [\n");
	for(int i=0; i<xSize; ++i)
	{
		for(int j=0; j<ySize; ++j)
		{
			for(int k=0; k<zSize; ++k)
			{
				if( !(i == 0 && j == 0 && k == 0) ) printf(",\n");
				char *val = ( map[k*(xSize*ySize)+j*xSize+i] ) ? "true" : "false";
				printf("\t\t{\"x\":%d, \"y\":%d, \"z\":%d, \"value\":%s}", i, j, k, val);
			}
		}
	}
	printf("\n\t\t]\n\t}");
}

template <size_t xSize, size_t ySize, size_t zSize>
void countStats(unsigned char *oldMap, unsigned char *newMap, MapStats &stats)
{
	int oldAlive = 0, oldDead = 0;
	int newAlive = 0, newDead = 0;
	
	for(int k=0; k<zSize; ++k)
	{
		for(int j=0; j<ySize; ++j)
		{
			for(int i=0; i<xSize; ++i)
			{
				oldAlive += oldMap[k*(xSize*ySize)+j*xSize+i];
				newAlive += newMap[k*(xSize*ySize)+j*xSize+i];
			}
		}
	}
	stats.alive = newAlive;
	newDead= (xSize*ySize*zSize) - newAlive;
	oldDead = (xSize*ySize*zSize) - oldAlive;
	stats.dead = newDead;
	stats.births = (newAlive > oldAlive) ? newAlive - oldAlive : 0;
	stats.deaths = (newDead > oldDead) ? newDead - oldDead : 0;
}


